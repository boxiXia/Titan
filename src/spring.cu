#include "hip/hip_runtime.h"
//
// Created by Jacob Austin on 5/17/18.
//
#include "spring.h"
#include <cmath>

using namespace titan;

namespace titan {

Vec Spring::getForce() { // computes force on right object. left force is - right force.
  //    Vec temp = (_right -> pos) - (_left -> pos);
  //    return _k * (_rest - temp.norm()) * (temp / temp.norm());

    Vec temp = (_left -> pos) - (_right -> pos);
    Vec spring_force = _k * (temp.norm() - _rest) * (temp / temp.norm());

    spring_force += dot(_left -> vel - _right -> vel, temp / temp.norm()) * _damping * (temp / temp.norm());

    return spring_force;
}

void Spring::setForce() { // computes force on right object. left force is - right force.
    Vec f = getForce();
    _right -> force += f;
    _left -> force += -f;
}

// this function is currently unneeded because springs do not need to be updated.
// this should be updated if any features are implemented to change spring parameters.
void Spring::update(const CUDA_SPRING & spr) {}

void Spring::defaultLength() { _rest = (_left -> pos - _right -> pos).norm() ; } // sets rest length

void Spring::setLeft(Mass * left) {
    if (_left) {
        _left -> decrementRefCount();
    }

    _left = left;
    _left -> ref_count++;

} // sets left mass (attaches spring to mass 1)

void Spring::setRight(Mass * right) {
    if (_right) {
        _right -> decrementRefCount();
    }

    _right = right;
    _right -> ref_count++;
}

CUDA_SPRING::CUDA_SPRING(const Spring & s) {
    _left = (s._left == nullptr) ? nullptr : s._left -> arrayptr;
    _right = (s._right == nullptr) ? nullptr : s. _right -> arrayptr;
    _k = s._k;
    _rest = s._rest;
    _type = s._type;
    _omega = s._omega;
    _damping = s._damping;
}

CUDA_SPRING::CUDA_SPRING(const Spring & s, CUDA_MASS * left, CUDA_MASS * right) {
    _left = left;
    _right = right;
    _k = s._k;
    _rest = s._rest;
    _type = s._type;
    _omega = s._omega;
    _damping = s._damping;
}

} // namespace titan